#include "hip/hip_runtime.h"
#include "../utils/utils.h"
#include <hip/hip_runtime.h>
#include <stdio.h>


void printSrcSum(float *v, float *sum, const int N){
    for (int i = 0; i < N; i++) {
        printf("v[%i] = %f\n",i ,v[i]);
    }
    printf("v.sum() = %f\n", *sum);
}
void checkResult(float *hostRef, float *gpuRef) {
    printf("Results on Host Matches to Result on Device? %s", (*hostRef) == (*gpuRef) ? "true" : "false");
}

void initialData(float *ip, int size) {
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++) {
        ip[i] = (float) (rand() & 0xFF) / 10.0f;
    }

    return;
}

void sumElemArrayOnHost(float *v, float *hostRefSum, const int N) {
    for (int idx = 0; idx < N - 1; idx++) {
        (*hostRefSum)+=v[idx];
    }
}

__global__ void sumElemArrayOnGPU(float *v, const int N) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N){
        v[0]+=v[idx];
    }
}

int main() {
    // printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    ERR_SAFE(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    ERR_SAFE(hipSetDevice(dev));

    // set up data size of vectors
    // int nElem = 1 << 24;
    int nElem = 4;
    printf("Vector size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_v, *hostRefSum, *gpuRefSum;
    h_v = (float *) malloc(nBytes);

    hostRefSum = (float *) malloc(sizeof(float));
    gpuRefSum= (float *) malloc(sizeof(float));

    double iStart, iElaps;

    // initialize data at host side
    iStart = time_of_day_seconds();
    initialData(h_v, nElem);
    iElaps = time_of_day_seconds() - iStart;
    printf("initialData Time elapsed %f sec\n", iElaps);
    memset(hostRefSum, 0, nBytes);
    memset(gpuRefSum, 0, nBytes);
    printf("Host memory allocated %d bytes\n", (int)nBytes);

    // add vector at host side for result checks
    iStart = time_of_day_seconds();
    sumElemArrayOnHost(h_v, hostRefSum, nElem);
    iElaps = time_of_day_seconds() - iStart;
    printf("sumElemArrayOnHost Time elapsed %f sec\n", iElaps);

    // malloc device global memory
    float *d_v;
    printf("Device memory shall allocate %d bytes\n", (int)(nBytes));
    ERR_SAFE(hipMalloc(&d_v, nBytes));
    // CHECK(hipMalloc((float **) &d_v, nBytes));
    // CHECK(hipMalloc((float **) &d_sum, sizeof(float)));
    printf("Device memory allocated\n");


    // transfer data from host to device
    ERR_SAFE(hipMemcpy(d_v, h_v, nBytes, hipMemcpyHostToDevice));
    // CHECK(hipMemcpy(d_sum, gpuRefSum, sizeof(float), hipMemcpyHostToDevice));

    // invoke kernel at host side
    int iLen = nElem;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    iStart = time_of_day_seconds();
    sumElemArrayOnGPU<<<grid, block>>>(d_v, nElem);
    ERR_SAFE(hipDeviceSynchronize());
    iElaps = time_of_day_seconds() - iStart;
    printf("sumElemArrayOnGPU <<<  %d, %d  >>>  Time elapsed %f sec\n", grid.x,
           block.x, iElaps);

    // check kernel error
    ERR_SAFE(hipGetLastError());

    // copy kernel result back to host side
    ERR_SAFE(hipMemcpy(gpuRefSum, d_v, sizeof(float), hipMemcpyDeviceToHost));


    // print host result vs device result
    printf("\n\n\n");
    printf("Vector.sum() Host\n");
    printf("******************\n");
    printSrcSum(h_v, hostRefSum, nElem);
    printf("\n\n\n");
    printf("Vector.sum() Device\n");
    printf("******************\n");
    printSrcSum(h_v, gpuRefSum, nElem);
    printf("\n\n\n");
    // check results
    checkResult(hostRefSum, gpuRefSum);
    printf("\n\n\n");
    // free device global memory
    ERR_SAFE(hipFree(d_v));
    // CHECK(hipFree(d_sum));

    // free host memory
    free(h_v);
    free(hostRefSum);
    free(gpuRefSum);

    return (0);
}

