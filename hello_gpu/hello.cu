
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    printf("Hello from GPU! %d\n", threadIdx.x);
}

int main() {
    printf("Hello world from CPU!\n");
    helloFromGPU<<<1, 16>>>();
    hipDeviceSynchronize();
    return 0;
}
