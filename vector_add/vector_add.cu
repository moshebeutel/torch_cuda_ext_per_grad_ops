#include "hip/hip_runtime.h"
#include "../utils/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
                   gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");

    return;
}

void initialData(float *ip, int size) {
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++) {
        ip[i] = (float) (rand() & 0xFF) / 10.0f;
    }

    return;
}

void pairSortArraysOnHost(float *A, float *B, float *C, const int N) {
    for (int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void pairSortArrayOnGPU(float *A, float *B, float *C, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}

int main() {
    // printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    ERR_SAFE(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    ERR_SAFE(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 1 << 24;
    printf("Vector size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *) malloc(nBytes);
    h_B = (float *) malloc(nBytes);
    hostRef = (float *) malloc(nBytes);
    gpuRef = (float *) malloc(nBytes);

    double iStart, iElaps;

    // initialize data at host side
    iStart = seconds();
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    iElaps = seconds() - iStart;
    printf("initialData Time elapsed %f sec\n", iElaps);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add vector at host side for result checks
    iStart = seconds();
    pairSortArraysOnHost(h_A, h_B, hostRef, nElem);
    iElaps = seconds() - iStart;
    printf("sumArraysOnHost Time elapsed %f sec\n", iElaps);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    ERR_SAFE(hipMalloc((float **) &d_A, nBytes));
    ERR_SAFE(hipMalloc((float **) &d_B, nBytes));
    ERR_SAFE(hipMalloc((float **) &d_C, nBytes));

    // transfer data from host to device
    ERR_SAFE(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    ERR_SAFE(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    ERR_SAFE(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int iLen = 512;
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1) / block.x);

    iStart = seconds();
    pairSortArrayOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    ERR_SAFE(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("sumArraysOnGPU <<<  %d, %d  >>>  Time elapsed %f sec\n", grid.x,
           block.x, iElaps);

    // check kernel error
    ERR_SAFE(hipGetLastError());

    // copy kernel result back to host side
    ERR_SAFE(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    ERR_SAFE(hipFree(d_A));
    ERR_SAFE(hipFree(d_B));
    ERR_SAFE(hipFree(d_C));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return (0);
}

